#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define EPS2 0.67f
#define BSIZE 256
#define softeningSquared 0.025f
#define damping 0.999f

__device__  __shared__ float4 shPosition[256];

__device__ float3 calc_accel_body(float4 bi, float4 bj, float3 ai) {
	float3 r;
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;

	float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + EPS2;

	float distSixth = distSqr * distSqr * distSqr;
	float invDistCube = 1.0f / sqrtf(distSixth);

	float s = bj.w * invDistCube;

	ai.x += r.x * s;// * EPS2;
	ai.y += r.y * s;// * EPS2;
	ai.z += r.z * s;// * EPS2;
	return ai;
}

__device__ float3 calc_accel(float4 myPosition, float3 accel) {
	int i;
	extern __shared__ float4 shPosition[];

	for (i = 0; i < BSIZE; i++) {
		accel = calc_accel_body(myPosition, shPosition[i], accel);
	}
	return accel;
}

__global__ void galaxyKernel(float4 * pdata, unsigned int bodies, float step) {
	// shared memory
	extern __shared__ float4 shPosition[];

	// index of my body
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned int pLoc = y * gridDim.y * blockDim.y + x;
	unsigned int vLoc = bodies + pLoc;

	float4 myPosition = pdata[pLoc];
	float4 myVelocity = pdata[vLoc];

	float3 acc = { 0.0f, 0.0f, 0.0f };
	unsigned int loop = gridDim.x * gridDim.y;

	unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
	for (int i = 0; i < loop; i++) {
		shPosition[idx] = pdata[idx + BSIZE * i];
		__syncthreads();

		acc = calc_accel(myPosition, acc);
		__syncthreads();
	}

	// update velocity with above acc
	myVelocity.x += acc.x * step; // * 2.0f;
	myVelocity.y += acc.y * step; // * 2.0f;
	myVelocity.z += acc.z * step; // * 2.0f;

	myVelocity.x *= damping;
	myVelocity.y *= damping;
	myVelocity.z *= damping;

	// update position
	myPosition.x += myVelocity.x * step;
	myPosition.y += myVelocity.y * step;
	myPosition.z += myVelocity.z * step;

	__syncthreads();

	// update device memory
	pdata[pLoc] = myPosition;
	pdata[vLoc] = myVelocity;
}

extern "C" void cudaComputeGalaxy(float4 * pdata, int N, float step) {
	dim3 block(16, 16, 1);
	int dim = sqrt(N / 256);
	dim3 grid(dim ,dim ,1);

	grid.y = grid.y == 0 ? 1 : grid.y;
	grid.x = grid.x == 0 ? 1 : grid.x;

	galaxyKernel<<<grid, block>>>(pdata, N, step);

	hipDeviceSynchronize();
}
